
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

__global__ 
void vecAddkernel(float* A, float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] * B[i];
    }
    
}   

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // invocation

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}

int main() {
    std::ifstream ifs("./arrays.txt");
    std::ofstream ofs("./out.txt");
    int total;
    float *A, *B, *C;

    ifs >> total;
    A = new float[total];
    B = new float[total];
    C = new float[total];
    for (size_t i = 0; i < total; i++)
    {
        ifs >> A[i];
    }
    for (size_t i = 0; i < total; i++)
    {
        ifs >> B[i];
    }

    vecAdd(A, B, C, total);

    for (size_t i = 0; i < total; i++)
    {
        ofs << C[i] << ' ';
    }
    
    return 0;
}
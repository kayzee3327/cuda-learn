#include "hip/hip_runtime.h"
#include "vecAdd.h"

__global__ 
void vecAddkernel(float* A, float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] * B[i];
    }
}   

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // invocation
    vecAddkernel<<<(n + 256 - 1)/ 256, 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}
